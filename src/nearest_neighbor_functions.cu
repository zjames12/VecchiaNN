#include <omp.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <string>
//' GPU Error check function
//`
//' Kernels do not throw exceptions. They instead return exit codes. If the exit code is
//` not \code{cudaSuccess} an error message is printed and the code is aborted.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    /*printf(cudaGetErrorString(code));
    printf("\n");*/
    if (code != hipSuccess)
    {
        // printf("fail%i\n", code);
        printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);

        //if (abort) exit(code);
    }

}

#define INDEX(i, j) (i * (i + 1) / 2 + j)



// locs n * dim
// dist array of size n * (n + 1) / 2 representing a triangular matrix
template<typename T>
__global__ void calculate_distance_matrix(T* locs, T* dist, int* indicies, int n, int dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < j || i > n) {
        return;
    }
    T d = 0;
    T temp = 0;
    for (int k = 0; k < dim; k++) {
        temp = (locs[i * dim + k] - locs[j * dim + k]);
        d += temp * temp;
    }
    dist[INDEX(i, j)] = d;
    indicies[INDEX(i, j)] = j + 1;
}

template<typename T>
__device__ int partition(T* dist, int* indicies, int left, int right, int row, int pivotIndex) {
    T pivotValue = dist[INDEX(row, pivotIndex)];
    dist[INDEX(row, pivotIndex)] = dist[INDEX(row, right)];
    dist[INDEX(row, right)] = pivotValue;
	
	int temp3 = indicies[INDEX(row, pivotIndex)];	
	indicies[INDEX(row, pivotIndex)] = indicies[INDEX(row, right)];
    indicies[INDEX(row, right)] = temp3;

	int storeIndex = left;
    for (int i = left; i < right; i++){
        if (dist[INDEX(row, i)] < pivotValue) {
            T temp = dist[INDEX(row, i)];
            dist[INDEX(row, i)] = dist[INDEX(row, storeIndex)];
            dist[INDEX(row, storeIndex)] = temp;

            int temp2 = indicies[INDEX(row, i)];
            indicies[INDEX(row, i)] = indicies[INDEX(row, storeIndex)];
            indicies[INDEX(row, storeIndex)] = temp2;
			storeIndex++;
        }
    }
    T temp = dist[INDEX(row, right)];
    dist[INDEX(row, right)] = dist[INDEX(row, storeIndex)];
    dist[INDEX(row, storeIndex)] = temp;

    int temp2 = indicies[INDEX(row, right)];
    indicies[INDEX(row, right)] = indicies[INDEX(row, storeIndex)];
    indicies[INDEX(row, storeIndex)] = temp2;
    return storeIndex;
}

template<typename T>
__device__ int hoare_partition(T* dist, int* indicies, int lo, int hi, int row, int pivotIndex) {
  T pivot = dist[INDEX(row, (hi - lo)/2 + lo)];
  int i = lo - 1;
  int j = hi + 1;
  while (i < j) {
    do {
      i++;
    } while (dist[INDEX(row, i)] < pivot);
    do {
      j--;
    }
    while (dist[INDEX(row, j)] > pivot);

    if (i >= j){
      return j;
    }
    T temp = dist[INDEX(row, i)];
    dist[INDEX(row, i)] = dist[INDEX(row, j)];
    dist[INDEX(row, j)] = temp;

    int temp2 = indicies[INDEX(row, i)];
    indicies[INDEX(row, i)] = indicies[INDEX(row, j)];
    indicies[INDEX(row, j)] = temp2;
  }
  return -1;
}

template<typename T>
__device__ void select(T* dist, int* indicies, int row, int left, int right, int k){
    int pivotIndex;
    while (left < right) {
        pivotIndex = (left + right) / 2;
        pivotIndex = hoare_partition(dist, indicies, left, right, row, pivotIndex);
        if (k == pivotIndex) {
            return;
        } else if (k < pivotIndex) {
            right = pivotIndex;
        } else {
            left = pivotIndex + 1;
        }
    }
}

template<typename T>
__global__ void sort_distance_matrix(T* dist, int* indicies, int n, int m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n || i < m + 1) {
        return;
    }
    select(dist, indicies, i, 0, i, m);
}

__global__ void create_nn_array(int* indicies, int* NNarray, int n, int m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= n || j > m || i < j){
        return;
    }
    NNarray[i * (m + 1) + j] = indicies[INDEX(i, j)];
}

extern "C"
int* nearest_neighbors(double* locs, int m, int n, int dim) {

    double *d_locs, *d_dist;
	int *d_NNarray;
	//T *d_NNarray;
	int* d_indicies;

    gpuErrchk(hipMalloc((void**)&d_locs, sizeof(double) * n * dim));
    gpuErrchk(hipMalloc((void**)&d_NNarray, sizeof(int) * n * (m + 1)));
    gpuErrchk(hipMemcpy(d_locs, locs, sizeof(double) * n * dim, hipMemcpyHostToDevice));
    
    
    gpuErrchk(hipMalloc((void**)&d_indicies, sizeof(int) * n * (n + 1) / 2));
    gpuErrchk(hipMalloc((void**)&d_dist, sizeof(double) * n * (n + 1) / 2));

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((n + threadsPerBlock.x -1) / threadsPerBlock.x, (n+threadsPerBlock.y -1) / threadsPerBlock.y);
    calculate_distance_matrix<<<numBlocks,threadsPerBlock>>>(d_locs, d_dist, d_indicies, n, dim);
    hipDeviceSynchronize();

    // dim3 threadsPerBlock2(32,1);
    // dim3 numBlocks2((n + 32 - 1) / 32, 1);
    int threadsPerBlock2 = 32;
    int numBlocks2 = (n + 32 - 1) / 32;
    sort_distance_matrix<<<numBlocks2, threadsPerBlock2>>>(d_dist, d_indicies, n, m);
    hipDeviceSynchronize();

    dim3 threadsPerBlock3(32, m + 1);
    create_nn_array<<<numBlocks2 , threadsPerBlock3>>>(d_indicies, d_NNarray, n, m);
    // create_nn_array2<<<numBlocks2 , threadsPerBlock3>>>(d_dist, d_NNarray, n, m);
    hipDeviceSynchronize();
    
    int* NNarray = (int*) malloc(sizeof(int) * n * (m + 1));
    gpuErrchk(hipMemcpy(NNarray, d_NNarray, sizeof(int) * n * (m + 1), hipMemcpyDeviceToHost));
    
    // T* NNarray = (T*) malloc(sizeof(T) * n * (m + 1));
    // gpuErrchk(cudaMemcpy(NNarray, d_dist, sizeof(T) * n * (m + 1), cudaMemcpyDeviceToHost));
    
	return NNarray;
}

extern "C"
int* nearest_neighbors_single(float* locs, int m, int n, int dim) {

    float *d_locs, *d_dist;
	int *d_NNarray;
	//T *d_NNarray;
	int* d_indicies;

    gpuErrchk(hipMalloc((void**)&d_locs, sizeof(float) * n * dim));
    gpuErrchk(hipMalloc((void**)&d_NNarray, sizeof(int) * n * (m + 1)));
    gpuErrchk(hipMemcpy(d_locs, locs, sizeof(float) * n * dim, hipMemcpyHostToDevice));
    
    
    gpuErrchk(hipMalloc((void**)&d_indicies, sizeof(int) * n * (n + 1) / 2));
    gpuErrchk(hipMalloc((void**)&d_dist, sizeof(float) * n * (n + 1) / 2));

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((n + threadsPerBlock.x -1) / threadsPerBlock.x, (n+threadsPerBlock.y -1) / threadsPerBlock.y);
    calculate_distance_matrix<<<numBlocks,threadsPerBlock>>>(d_locs, d_dist, d_indicies, n, dim);
    hipDeviceSynchronize();

    // dim3 threadsPerBlock2(32,1);
    // dim3 numBlocks2((n + 32 - 1) / 32, 1);
    int threadsPerBlock2 = 32;
    int numBlocks2 = (n + 32 - 1) / 32;
    sort_distance_matrix<<<numBlocks2, threadsPerBlock2>>>(d_dist, d_indicies, n, m);
    hipDeviceSynchronize();

    dim3 threadsPerBlock3(32, m + 1);
    create_nn_array<<<numBlocks2 , threadsPerBlock3>>>(d_indicies, d_NNarray, n, m);
    // create_nn_array2<<<numBlocks2 , threadsPerBlock3>>>(d_dist, d_NNarray, n, m);
    hipDeviceSynchronize();
    
    int* NNarray = (int*) malloc(sizeof(int) * n * (m + 1));
    gpuErrchk(hipMemcpy(NNarray, d_NNarray, sizeof(int) * n * (m + 1), hipMemcpyDeviceToHost));
    
    // T* NNarray = (T*) malloc(sizeof(T) * n * (m + 1));
    // gpuErrchk(cudaMemcpy(NNarray, d_dist, sizeof(T) * n * (m + 1), cudaMemcpyDeviceToHost));
    
	return NNarray;
}